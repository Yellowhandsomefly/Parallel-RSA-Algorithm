
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>


char buffer[1024];
const int MAX_DIGITS = 50;
int i,j = 0;

struct public_key_class{
  long long modulus;
  long long exponent;
};

struct private_key_class{
  long long modulus;
  long long exponent;
};


// This should totally be in the math library.
long long gcd(long long a, long long b)
{
  long long c;
  while ( a != 0 ) {
    c = a; a = b%a;  b = c;
  }
  return b;
}


long long ExtEuclid(long long a, long long b)
{
 long long x = 0, y = 1, u = 1, v = 0, gcd = b, m, n, q, r;
 while (a!=0) {
   q = gcd/a; r = gcd % a;
   m = x-u*q; n = y-v*q;
   gcd = a; a = r; x = u; y = v; u = m; v = n;
   }
   return y;
}

__global__ void rsa_modExp(long long b, long long e, long long m, long long* array)
{
  if (b < 0 || e < 0 || m <= 0){
    printf("error arguments.\n");
  }

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int thread_nums = gridDim.x * blockDim.x;

  long long en_result = 1;
 
	long long i;
	for(i = index; i < e ; i+=thread_nums){
		//t_result = en_result % m;
		en_result = (en_result * b)% m;
  }
  if (index < e){
    array[index] = en_result;
  }
  else {
    array[index] = 0;
  }
	//return en_result;
	

}

// Calling this function will generate a public and private key and store them in the pointers
// it is given. 
void rsa_gen_keys(struct public_key_class *pub, struct private_key_class *priv, char *PRIME_SOURCE_FILE)
{
  FILE *primes_list;
  if(!(primes_list = fopen(PRIME_SOURCE_FILE, "r"))){
    fprintf(stderr, "Problem reading %s\n", PRIME_SOURCE_FILE);
    exit(1);
  }

  // count number of primes in the list
  long long prime_count = 0;
  do{
    int bytes_read = fread(buffer,1,sizeof(buffer)-1, primes_list);
    buffer[bytes_read] = '\0';
    for (i=0 ; buffer[i]; i++){
      if (buffer[i] == '\n'){
	prime_count++;
      }
    }
  }
  while(feof(primes_list) == 0);
  
  
  // choose random primes from the list, store them as p,q

  long long p = 0;
  long long q = 0;

  long long e = powl(2, 8) + 1;
  long long d = 0;
  char prime_buffer[MAX_DIGITS];
  long long max = 0;
  long long phi_max = 0;
  
  srand(time(NULL));
  
  do{
    // a and b are the positions of p and q in the list
    int a =  (double)rand() * (prime_count+1) / (RAND_MAX+1.0);
    int b =  (double)rand() * (prime_count+1) / (RAND_MAX+1.0);
    
    // here we find the prime at position a, store it as p
    rewind(primes_list);
    for(i=0; i < a + 1; i++){
    //  for(j=0; j < MAX_DIGITS; j++){
    //	prime_buffer[j] = 0;
    //  }
      fgets(prime_buffer,sizeof(prime_buffer)-1, primes_list);
    }
    p = atol(prime_buffer); 
    
    // here we find the prime at position b, store it as q
    rewind(primes_list);
    for(i=0; i < b + 1; i++){
      for(j=0; j < MAX_DIGITS; j++){
	prime_buffer[j] = 0;
      }
      fgets(prime_buffer,sizeof(prime_buffer)-1, primes_list);
    }
    q = atol(prime_buffer); 

    //here
    p = 8011;
    q = 8521;

    max = p*q;
    phi_max = (p-1)*(q-1);
  }
  while(!(p && q) || (p == q) || (gcd(phi_max, e) != 1));
 
  // Next, we need to choose a,b, so that a*max+b*e = gcd(max,e). We actually only need b
  // here, and in keeping with the usual notation of RSA we'll call it d. We'd also like 
  // to make sure we get a representation of d as positive, hence the while loop.
  d = ExtEuclid(phi_max,e);
  while(d < 0){
    d = d+phi_max;
  }

  printf("primes are %lld and %lld\n",(long long)p, (long long )q);
  // We now store the public / private keys in the appropriate structs

  //here
  d = 41956193;

  pub->modulus = max;
//pub->modulus = 2936519639;
  pub->exponent = e;
//pub->exponent = 257;
  priv->modulus = max;
//priv->modulus = 2936519639;
  priv->exponent = d;
  //priv->exponent = 1988060033;
}


long long *rsa_encrypt(const char *message, const unsigned long message_size, 
                     const struct public_key_class *pub)
{
  long long *encrypted = (long long *)malloc(sizeof(long long)*message_size);
  if(encrypted == NULL){
    fprintf(stderr,
     "Error: Heap allocation failed.\n");
    return NULL;
  }

  long long i = 0;
  long long j = 0;
  dim3 dimBlock(512);
  dim3 dimGrid(16);

  int size = 512 * 16 * sizeof(long long);
  long long *array;
  long long host_array[size];
  hipMalloc((void**)&array, size);
  long long result = 1;

  for(i=0; i < message_size; i++){
    rsa_modExp<<<dimGrid, dimBlock>>>(message[i], pub->exponent, pub->modulus, array);

    hipMemcpy(host_array, array, size, hipMemcpyDeviceToHost);

    result = 1;
    for(j=0; j < 512 * 16; j++){
      if (j < pub->exponent) {
        result = (result * host_array[j]) % pub->modulus;
      }
    }
    
    encrypted[i] = result;
  }
  return encrypted;
}


char *rsa_decrypt(const long long *message, 
                  const unsigned long message_size, 
                  const struct private_key_class *priv)
{
  if(message_size % sizeof(long long) != 0){
    fprintf(stderr,
     "Error: message_size is not divisible by %d, so cannot be output of rsa_encrypt\n", (int)sizeof(long long));
     return NULL;
  }
  // We allocate space to do the decryption (temp) and space for the output as a char array
  // (decrypted)
  char *decrypted = (char *)malloc(message_size/sizeof(long long));
  char *temp = (char *)malloc(message_size);
  if((decrypted == NULL) || (temp == NULL)){
    fprintf(stderr,
     "Error: Heap allocation failed.\n");
    return NULL;
  }
  // Now we go through each 8-byte chunk and decrypt it.
  long long i = 0;
  long long j = 0;
  dim3 dimBlock(512);
  dim3 dimGrid(16);

  int size = 512 * 16 * sizeof(long long);
  long long *array;
  long long host_array[size];

  hipMalloc((void**)&array, size);
  long long result = 1;

  for(i=0; i < message_size/8; i++){
    rsa_modExp<<<dimGrid, dimBlock>>>(message[i], priv->exponent, priv->modulus, array);

    hipMemcpy(host_array, array, size, hipMemcpyDeviceToHost);

    result = 1;
    for(j=0; j < 512 * 16 ; j++){
      if (j < priv->exponent) {
        result = (result * host_array[j]) % priv->modulus;
      }
    }

    temp[i] = result;
  }

  // The result should be a number in the char range, which gives back the original byte.
  // We put that into decrypted, then return.
  for(i=0; i < message_size/8; i++){
    decrypted[i] = temp[i];
  }
  free(temp);
  return decrypted;
}

int main(int argc, char **argv)
{
  char *PRIME_SOURCE_FILE = "primes.txt";
  struct public_key_class pub[1];
  struct private_key_class priv[1];
  rsa_gen_keys(pub, priv, PRIME_SOURCE_FILE);

  printf("Private Key:\n Modulus: %lld\n Exponent: %lld\n", (long long)priv->modulus, (long long) priv->exponent);
  printf("Public Key:\n Modulus: %lld\n Exponent: %lld\n", (long long)pub->modulus, (long long) pub->exponent);
  
  FILE* fptr = fopen("test.txt", "r");
  if(fptr == NULL){
	  printf("e1\n");
  }
  int filesize = fseek(fptr, 0, SEEK_END);
  char* buff = (char*)malloc(100 * sizeof(char));
  fseek(fptr, 0, SEEK_SET);
  int n = fread(buff, 100, 1, fptr);
  
  char message[101];
  strncpy(message, buff, sizeof(message)-1); 
  message[100] = '\0';

  int i;

  printf("Original:\n");
  //for(i=0; i < strlen(message); i++){
    //printf("%lld\n", (long long)message[i]);
	printf("%s\n", message);
  //}  
  
  long long *encrypted = rsa_encrypt(message, sizeof(message), pub);
  if (!encrypted){
    fprintf(stderr, "Error in encryption!\n");
    return 1;
  }
  printf("Encrypted:\n");
  for(i=0; i < strlen(message); i++){
    printf("%lld\n", (long long)encrypted[i]);
  }  
  
  char *decrypted = rsa_decrypt(encrypted, 8*sizeof(message), priv);
  if (!decrypted){
    fprintf(stderr, "Error in decryption!\n");
    return 1;
  }
  printf("Decrypted:\n");
  //for(i=0; i < strlen(message); i++){
    //printf("%lld\n", i, (long long)decrypted[i]);
	//printf("%s\n", decrypted);
  //}  
  
  FILE* fptr1 = fopen(argv[argc - 1], "w");
  fwrite(decrypted, 100, 1, fptr1);
  
  printf("\n");
  free(encrypted);
  free(decrypted);
  free(buff);
  fclose(fptr1);
  fclose(fptr);
  return 0;
}
